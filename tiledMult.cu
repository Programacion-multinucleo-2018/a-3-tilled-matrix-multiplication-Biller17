#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <chrono>

using namespace std;

#define TILEDIM 8


//Code used from examples and modified for activity
//Adrian Biller A01018940
//matrix multiplication with 2D 2D using tiled method





//inicialization of matrices
void initialData(int *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX/10));
    }

    return;
}

//printing arrays
void printArray(int * arr, int size)
{
  int totalSize = size * size;
  int row = 1;
  for(int x = 0; x < totalSize; x++){
    printf("%d ", arr[x]);
    if((size * row)-1 == x){
      row++;
      printf("\n");
    }
  }
}


//multiplication of matrices using cpu
void multiplyMatrixOnHost(int *A, int *B, int *C, const int nx,
                     const int ny)
{
      for(int i = 0; i < nx; i++){
        for(int j = 0; j < nx ; j++){
          for(int k = 0; k < nx; k++){
            C[i*nx+j] += A[i*nx+k] * B[k*nx+j];
          }
        }
      }

    return;
}

//checking result of gpu and comparing them with cpu matrix
void checkResult(int *hostRef, int *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}




//matrix calculation using cpu
__global__ void multMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx,
    int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;


    if (ix < nx && iy < ny){
        for(int k = 0; k < nx; k++){
          MatC[ix * nx + iy] += MatA[ix * nx + k] * MatB[k * nx + iy];
        }
    }
}

//matrix calculation using tile method
__global__ void tiledMult(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // unsigned int idx = iy * nx + ix;

    __shared__ float sharedMatA[TILEDIM][TILEDIM];
    __shared__ float sharedMatB[TILEDIM][TILEDIM];

    printf("%d\n", blockDim.x);
    printf("%d\n", TILEDIM);
    // for(int i = 0; i < DIM; i ++) {
    //   for(int j = 0; j < DIM; j++) {
    //     sharedMatA[i][j] = 0;
    //     sharedMatB[i][j] = 0;
    //   }
    // }
}



int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    // int nx = 1 << 12;
    // int ny = 1 << 12;
    int nx = 4000;
    int ny = 4000;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    int *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (int *)malloc(nBytes);
    h_B = (int *)malloc(nBytes);
    hostRef = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    // initialize data at host side

    initialData(h_A, nxy);
    initialData(h_B, nxy);
    // printArray(h_A, nx);
    // printf("\n");
    // printArray(h_B, nx);
    // printf("\n");

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result SAFE_CALLs
    auto start_cpu =  chrono::high_resolution_clock::now();
    multiplyMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    auto end_cpu =  chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multiplyMatrixOnHost elapsed %f ms\n", duration_ms.count());

    // malloc device global memory
    int *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");
    SAFE_CALL(hipMemset(d_MatC, 0, nBytes), "Error setting d_MatC to zeros");

    // invoke kernel at host side
    int dimx = TILEDIM;
    int dimy = TILEDIM;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // start_cpu =  chrono::high_resolution_clock::now();
    // multMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    // end_cpu =  chrono::high_resolution_clock::now();
    //
    // duration_ms = end_cpu - start_cpu;
    //
    // printf("sumMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
    //        grid.y,
    //        block.x, block.y, duration_ms.count());
    //
    // // SAFE_CALL kernel error
    // SAFE_CALL(hipGetLastError(), "Error with last error");
    //
    // // copy kernel result back to host side
    // SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // printArray(hostRef, nx);
    // printf("Host\n");
    // printArray(gpuRef, nx);
    // printf("GPU\n");
    // // check device results
    // checkResult(hostRef, gpuRef, nxy);


    // dimx = TILEDIM;
    // dimy = TILEDIM;
    // dim3 block(dimx, dimy);
    // dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    //calculating matrix multiplication using Tiling
    start_cpu =  chrono::high_resolution_clock::now();
    tiledMult<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  chrono::high_resolution_clock::now();

    duration_ms = end_cpu - start_cpu;

    printf("Matrix multiplication with tiling <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
           grid.y,
           block.x, block.y, duration_ms.count());


    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");


    checkResult(hostRef, gpuRef, nxy);




    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}
